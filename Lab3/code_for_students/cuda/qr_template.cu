//
//  main.c
//  qr
//
//  Created by Zia Ul-Huda on 21/11/2016.
//  Copyright © 2016 TU Darmstadt. All rights reserved.
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

void showGPUMem();

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                file, line, hipGetErrorString( err ) );
	showGPUMem();
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    /*   err = cudaDeviceSynchronize();
     if( cudaSuccess != err )
     {
     fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
     file, line, cudaGetErrorString( err ) );
     exit( -1 );
     }*/
#endif

    return;
}


typedef struct {
    int m, n;
    double * v;
} mat_t, mat;

#define BLOCK_SIZE 16
#define MAX_INT 100
#define EPSILON 0.00000001

int numBlocks;
dim3 dimGrid, dimBlock;
int numBlocksSingle, numThreadsSingle;

//get current wall time
double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        exit(-1);
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

//creates a new structure of mat type with m*n dimensions and
//returns its pointer
mat* matrix_new(int m, int n)
{
    mat *x = (mat*)malloc(sizeof(mat_t));
    x->v = (double*)calloc(sizeof(double), m * n);
    x->m = m;
    x->n = n;
    return x;
}

/**
 * Creates a new structure of type mat
 * on the device and initializes it. It returns
 * the pointer to the structure in *x
 */
void cuda_matrix_new(int m, int n, mat** x)
{
    double* d_arr;
	mat* temp=(mat_t*)malloc(sizeof(mat_t)) ;
	temp->m = m;
        temp->n = n;
	
    //allocate mat struct on device
	hipMalloc(x,sizeof(mat_t));

    CudaCheckError();

    //allocate array on device and set it to 0

    hipMalloc(&d_arr, m*n*sizeof(double));

    CudaCheckError();
    hipMemset(d_arr, 0, sizeof(double) * m * n);
    CudaCheckError();

    //store the device pointer in temp object
    temp->v = d_arr;
   
    //copy the temp to device object
	fprintf(stderr,"*x: %p and &temp:%p\n",*x,temp->v);
    hipMemcpy(*x, temp, sizeof(mat_t),
				hipMemcpyHostToDevice);

    CudaCheckError();
free(temp);
}

//delete a matrix
void matrix_delete(mat *m)
{
    free(m->v);
    free(m);
}

/**
 * Free the memory of the structure pointed to by
 * m on the device. Make sure to also free the memory
 * of the elements of the matrix.
 */
void cuda_matrix_delete(mat *m)
{
    mat* temp;

    // Copy m to host

	hipMemcpy(&temp,m,sizeof(mat),hipMemcpyDeviceToHost);

    CudaCheckError();

    // Free array in m

	hipFree(temp->v);

    CudaCheckError();

    // Free m
    hipFree(m);
    CudaCheckError();

}

//calculate transpose of a matrix
void matrix_transpose(mat *m)
{
    int i,j;
    for (i = 0; i < m->m; i++) {
        for (j = 0; j < i; j++) {
            double t = m->v[i*m->n+j];
            m->v[i*m->n+j] = m->v[j*m->m+i];
            m->v[j*m->m+i] = t;
        }
    }
}

/**
 * Transpose the matrix on the device
 */
__global__
void cuda_matrix_transpose(mat* m){
    //Calculate the row of current element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    //Calculate the column of current element
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    //Just the threads in lower triangle should swap m elements
    if(row<m->m && col<m->n && row<col){
        double t = m->v[row*m->n+col];
        m->v[row*m->n+col] = m->v[col*m->m+row];
        // Finish swapping
       	m->v[col*m->m+row] = t;
    }
}

//Create a new matrix and initialize its elements randomly
mat* matrix_create(int m, int n)
{
    mat *x = matrix_new(m, n);
    srand(time(NULL));
    int i,j;
    for (i = 0; i < m*n; i++){
        j=rand() % MAX_INT;
        x->v[i] = j;
    }
    return x;
}

//multiplication of two matrixes
mat* matrix_mul(mat *x, mat *y)
{
    if (x->n != y->m) return NULL;
    mat *r = matrix_new(x->m, y->n);
    int i,j,k;
    for (i = 0; i < x->m; i++)
        for (j = 0; j < y->n; j++)
            for (k = 0; k < x->n; k++)
                r->v[i*r->n+j] += x->v[i*x->n+k] * y->v[k*y->n+j];
    return r;
}


/**
 * Multiply matrices x and y on the device and store
 * the result in r on the device. r contains already
 * enough memory for the result matrix.
 */
__global__
void cuda_matrix_mul(mat* x, mat* y, mat* r)
{
    //calculate the row and column index of matrixes x and y respectively
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < x->m && col < y->n){
        double rValue=0;

        //each thread computes one element of r
        int k;
        for(k=0; k < x->n; ++k)
            rValue += x->v[row*x->n+k]*y->v[k*y->n+col];

        r->v[row*r->n+col] = rValue;
    }
}

//calculate minor of a matrix given int d. Set first d
//diagonal entries to 1 and and set the rest of elements of
//first d rows and columns to zero. Then copy rest of the
//elements from the given matrix and return the pointer to new
//object
mat* matrix_minor(mat *x, int d)
{
    mat *m = matrix_new(x->m, x->n);
    int i,j;
    for (i = 0; i < d; i++)
        m->v[i*m->n+i] = 1;
    for (i = d; i < x->m; i++)
        for (j = d; j < x->n; j++)
            m->v[i*m->n+j] = x->v[i*x->n+j];
    return m;
}

/**
 * Calculate minor of a matrix given int d on device
 */
__global__
void cuda_matrix_minor(mat* x, int d, mat* m){
    //calculate the row and column index of matrixes x and y
    //respectively
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < x->m && col < x->n){
        if (row == col && row < d)
            m->v[row*m->n+col]=1;
        if(row >= d && row < x->m && col >= d && col < x->n)
            m->v[row*m->n+col]=x->v[row*x->n+col];
    }
}

// c = a + b * s
double *vmadd(double a[], double b[], double s, double c[], int n)
{
    int i;
    for (i = 0; i < n; i++)
        c[i] = a[i] + s * b[i];
    return c;
}

/**
 * c = a + b * s on device
 */
__global__
void cuda_vmadd(double a[], double b[], double *s, double c[], int n){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row <n)
        c[row] = a[row] + b[row]*(*s);
}

// m = I - 2vv^T
mat* vmul(double v[], int n)
{
    mat *x = matrix_new(n, n);
    int i,j;
    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++)
            x->v[i*x->n+j] = -2 *  v[i] * v[j];
    for (i = 0; i < n; i++)
        x->v[i*x->n+i] += 1;

    return x;
}

/**
 * m = I - 2vv^T  on device
 */
__global__
void cuda_vmul(double v[], int n, mat* m)
{
    //calculate the row and column index of matrixes x and y respectively
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < n && col < n){
        m->v[row*m->n+col] = -2*v[row]*v[col];
        if(row == col)
            m->v[row*m->n+col] += 1;
    }

}

// ||x||
double vnorm(double x[], int n)
{
    double sum = 0;
    int i;
    for (i = 0; i < n; i++) sum += x[i] * x[i];
    return sqrt(sum);
}


/**
 * Call with <<1,1>>
 * ||x|| on device and result is given in *a.
 * If flag is true (!= 0) a is multiplied with -1
 */
__global__
void cuda_vnorm(double x[], int n, double *a, int flag)
{
    if(blockIdx.x == 0 && threadIdx.x == 0 ) {
        double sum = 0;
        int i;
        for (i = 0; i < n; i++)
            sum += x[i]*x[i];
        *a = sqrt(sum);
        if (flag) *a = *a*(-1);
    }
}


// y = x / d
double* vdiv(double x[], double d, double y[], int n)
{
    int i;
    for (i = 0; i < n; i++) y[i] = x[i] / d;
    return y;
}

/**
 * y = x / d on device
 */
__global__
void cuda_vdiv(double x[], double *d, double y[], int n)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row <n)
       y[row]=x[row]/(*d);
}

// take c-th column of m, put in v
double* mcol(mat *m, double *v, int c)
{
    int i;
    for (i = 0; i < m->m; i++)
        v[i] = m->v[i*m->n+c];
    return v;
}

/**
 * Take c-th column of m, put in v on device
 */
__global__
void cuda_mcol(mat *m, double *v, int c)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < m->m)
        v[row] = m->v[row*m->n+c];
}

/**
 * Initialize vector e where k-th element is set to 1
 * and all other are 0 on device
 */
__global__
void cuda_initialize_e(double* e, int n, int k){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < n){
        if(row=k){
            e[row] = 1;
		}else{
            e[row] = 0;
		}
    }
}

//visualize a matrix
void matrix_show(mat *m)
{
    int i,j;
    for(i = 0; i < m->m; i++) {
        for (j = 0; j < m->n; j++) {
            printf(" %8.3f", m->v[i*m->n+j]);
        }
        printf("\n");
    }
    printf("\n");
}

//householder calculations
void householder(mat *m, mat **R, mat **Q)
{
    mat *q[m->m];
    mat *z = m, *z1;
    int i,k;
    for (k = 0; k < m->n && k < m->m - 1; k++) {
        double e[m->m], x[m->m], a;
        z1 = matrix_minor(z, k);
        if (z != m) matrix_delete(z);
        z = z1;

        mcol(z, x, k);
        a = vnorm(x, m->m);
        if (m->v[k*m->n+k] > 0) a = -a;

        for (i = 0; i < m->m; i++)
            e[i] = (i == k) ? 1 : 0;

        vmadd(x, e, a, e, m->m);
        vdiv(e, vnorm(e, m->m), e, m->m);
        q[k] = vmul(e, m->m);
        z1 = matrix_mul(q[k], z);
        if (z != m) matrix_delete(z);
        z = z1;
    }
    matrix_delete(z);
    *Q = q[0];
    *R = matrix_mul(q[0], m);
    for (i = 1; i < m->n && i < m->m - 1; i++) {
        z1 = matrix_mul(q[i], *Q);
        if (i > 1) matrix_delete(*Q);
        *Q = z1;
        matrix_delete(q[i]);
    }
    matrix_delete(q[0]);
    z = matrix_mul(*Q, m);
    matrix_delete(*R);
    *R = z;
    matrix_transpose(*Q);
}

/**
 * Householder calculations with calls to device kernels
 */
void cuda_householder(mat *m, mat **R, mat **Q, mat *original)
{
    mat *q;
    mat *z = m, *z1;
    int k;
    double *e, *x, *a;

    // Alloc vector e
    hipMalloc((void**)&e, sizeof(double) * original->m);
    CudaCheckError();

    // Alloc vector x
    hipMalloc((void**)&x, sizeof(double) * original->m);
    CudaCheckError();

    // Alloc scalar a
    hipMalloc((void**)&a, sizeof(double));
    CudaCheckError();

    //showGPUMem();

    for (k = 0; k < original->n && k < original->m - 1; k++) {

        // Allocate and init matrix z1
	fprintf(stderr,"here lays the matrix z1: %p",&z1);

        cuda_matrix_new(original->m,original->n, &z1);
	fprintf(stderr, "MatrixNew\n");

        // One thread calculates one element of matrix z1
        cuda_matrix_minor<<<dimGrid, dimBlock>>>(original, k, z1 ); //Versuch, Idee alternativ (original, k, z1) wenn original->v == m->v
		fprintf(stderr, "matrix_minor\n");
        CudaCheckError();
        if (z != m) cuda_matrix_delete(z);
        z = z1;

        // One thread calculates one element of vector x
        cuda_mcol<<<numBlocksSingle,numThreadsSingle>>>(z, x, k); //Abgeschrieben von sequentieller Funktion
        //z müsste ein Möglichkeit bieten auf z->v bzw m->v zuzugreifen
		fprintf(stderr, "mcol\n");
        CudaCheckError();

        int f = (original->v[k*original->n+k] > 0) ? 1 : 0;
        // Call cuda_vnorm with only one thread
        cuda_vnorm<<<1,1>>>(x, original->m, a, f); //Eingetragen: 1,1
		fprintf(stderr, "vnorm\n");
        CudaCheckError();

        // One thread calculates one element of vector e
        cuda_initialize_e<<<numBlocksSingle,numThreadsSingle>>>(e, original->m, k); //Eingetragen (e, original->m, k)
		fprintf(stderr, "cuda_initalize\n");
        CudaCheckError();

        // One thread calculates one element of vector e
        cuda_vmadd<<<1,1>>>(x, e, a, e, original->m); //Eingetragen 1,1 und original->m
		fprintf(stderr, "vmadd\n");
        CudaCheckError();

        // Call cuda_vnorm with only one thread
        cuda_vnorm<<<1,1>>>(e, original->m, a, 0);
        CudaCheckError();
        // One thread calculates one element of vector e with cuda_vdiv
        cuda_vdiv<<<1,1>>>(e, a, e, original->m); //EIngetragen: cuda_cdiv und 1,1
		fprintf(stderr, "vdiv\n");
        CudaCheckError();

        // Allocate matrix q
	fprintf(stderr,"here lays the matrix q: %p",&q);
        cuda_matrix_new(original->m, original->n, &q);
        // One thread calculates one element of matrix q
        cuda_vmul<<<dimGrid, dimBlock>>>(e, original->m, q);
	fprintf(stderr, "vmul\n");
        CudaCheckError();

        // Allocate matrix z1
        cuda_matrix_new(original->m,original->n,&z1);
        // One thread calculates one element of matrix z1
        // Calculate matrix product z1 = q*z with cuda_matrix_mul
        cuda_matrix_mul<<<numBlocksSingle,numThreadsSingle>>>(q,z,z1); //Komplett selbst geschrieben
	fprintf(stderr, "cuda_matrix_mul\n");
        CudaCheckError();

        if (z != m) cuda_matrix_delete(z);
        z = z1;

        if(k==0){
            *Q = q;
        }
        else if(k>0){
            cuda_matrix_new(original->m, original->m, &z1);
            cuda_matrix_mul<<<dimGrid, dimBlock>>>(q, *Q, z1);
            CudaCheckError();

            cuda_matrix_delete(*Q);
            *Q = z1;
            cuda_matrix_delete(q);
        }

    }

    // Free temporary storage on device
    hipFree(e);
    CudaCheckError();
    hipFree(x);
    CudaCheckError();
    hipFree(a);
    CudaCheckError();
    cuda_matrix_delete(z);


    cuda_matrix_new(original->m, original->n, R);
    // Result matrix R
    cuda_matrix_mul<<<dimGrid, dimBlock>>>(*Q, m, *R);
    CudaCheckError();
    // Result matrix Q
    cuda_matrix_transpose<<<dimGrid, dimBlock>>>(*Q);
    CudaCheckError();
}

/** Task2
 * Deep copy of matrix x to the device.
 * Return pointer to new structure on device in *dX
 */
void copyToDevice(mat** dX, mat* x){
    mat temp;
    temp.m = x->m;
    temp.n = x->n;
    double* d_arr;

    //allocate device matrix
    hipMalloc((void**)dX, sizeof(mat));
    CudaCheckError();

    //allocate device array
    hipMalloc((void**)&d_arr, x->m*x->n*sizeof(double));
	CudaCheckError();

    //copy contents of x array
    hipMemcpy(d_arr, x->v, x->m*x->n*sizeof(double), hipMemcpyHostToDevice);
    CudaCheckError();

    //save d_arr in temp
    temp.v = d_arr;

    //copy the temp to device object
    hipMemcpy(*dX, &temp, sizeof(mat_t), hipMemcpyHostToDevice);
    CudaCheckError();
}

/**
 * Deep copy of matrix dX to the host.
 * Return pointer to new structure on host in *x
 */
void copyToHost(mat** x, mat* dX){
    *x = (mat*)malloc(sizeof(mat_t));
    hipMemcpy(*x, dX, sizeof(mat_t), hipMemcpyDeviceToHost);
    CudaCheckError();

    double* temp = (double*)malloc(sizeof(double) * (*x)->m * (*x)->n);
    // Copy array of dX to temp
    hipMemcpy(temp, (*x)->v, sizeof(double) * (*x)->m * (*x)->n, hipMemcpyDeviceToHost);
    CudaCheckError();

    (*x)->v = temp;
}

//check if two matrixes are equal with their corrsponding element's values being within an epsilon
int is_equal(mat *m, mat *x){
    if(m->m != x->m || m->n != x->n) return 0;
    int i;

    for(i=0; i< (m->m * m->n); ++i)
        if(abs(m->v[i] - x->v[i]) > EPSILON) return 0;

    return 1;
}

void showGPUMem(){
    // show memory usage of GPU

    size_t free_byte ;
    size_t total_byte ;
    hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

    if ( hipSuccess != cuda_status ){
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
        exit(1);
    }

    double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;

    printf("GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n",
            used_db/1024.0/1024.0, free_db/1024.0/1024.0,
            total_db/1024.0/1024.0);
}

int main(int argc, char *argv[])
{
    if(argc != 3){
        puts("Usage: qr #rows #cols\n //#rows > 2 and #cols > 1\n");
        exit(0);
    }
    int row = atoi(argv[1]), col = atoi(argv[2]);

    if(row < 3 || col < 2){
        puts("Error: invalid number of rows or columns\n");
        exit(0);
    }

    int maxDim = (row > col) ? row : col;
    //use maxDim to calculate dimensions of grids and blocks for 2D cuda kernels
    numBlocks = maxDim / BLOCK_SIZE;
    if(maxDim % BLOCK_SIZE) numBlocks++;

    dimGrid.x = numBlocks; dimGrid.y = numBlocks;
    // Every CUDA block is of size (x,y,z) = (BLOCK_SIZE,BLOCK_SIZE,1) threads
    dimBlock.x = BLOCK_SIZE; dimBlock.y = BLOCK_SIZE;

    //dimensions of blocks and threads for 1D cuda kernels for vectors
    // Every CUDA block is of size (x,y,z) = (BLOCK_SIZE*BLOCK_SIZE,1,1)
    numThreadsSingle = BLOCK_SIZE * BLOCK_SIZE;
    numBlocksSingle = maxDim/numThreadsSingle;
    if(maxDim % numThreadsSingle) ++numBlocksSingle;

    mat *R = NULL, *Q = NULL, *dX = NULL, *dQ = NULL, *dR = NULL;
    //showGPUMem();
    //create a random row*col matrix
    mat *x = matrix_create(row, col);
    //puts("x"); matrix_show(x);
	fprintf(stderr,"matix x %d,%d,%p\n",x->m,x->n,x->v);

    double time_start = get_wall_time();

    //copy x to device
    copyToDevice(&dX, x);
	fprintf(stderr,"adress &X: %p\n",dX);
    //showGPUMem();
    //householder calculations on device
    cuda_householder(dX, &dR, &dQ, x);

    //copy the calculated dR and dQ to host
    copyToHost(&R, dR);
    copyToHost(&Q, dQ);

    double time_end = get_wall_time();

    //puts("Q"); matrix_show(Q);
    //puts("R"); matrix_show(R);

    // to show their product is the input matrix
    mat* dM = NULL;
    cuda_matrix_new(x->m, x->n, &dM);
    cuda_matrix_mul<<<dimGrid, dimBlock>>>(dQ, dR, dM);

    //copy resultant matrix to host
    mat* m = NULL;
    copyToHost(&m, dM);
    //puts("Q * R"); matrix_show(m);


    printf("Verification: ");
    if(is_equal(m, x))
        printf("Successful\n");
    else
        printf("Unsuccessful\n");

    printf("Time taken: %8.3f seconds\n",time_end - time_start);

    matrix_delete(x);
    matrix_delete(R);
    matrix_delete(Q);
    matrix_delete(m);
    cuda_matrix_delete(dX);
    cuda_matrix_delete(dQ);
    cuda_matrix_delete(dR);
    cuda_matrix_delete(dM);
	fflush(stdout);
    return 0;
}
